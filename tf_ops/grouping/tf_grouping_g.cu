
#include <hip/hip_runtime.h>
#include<stdio.h>


// input: radius (1), nsample (1), xyz1 (b,n,3), xyz2 (b,m,3)
// output: idx (b,m,nsample), pts_cnt (b,m)
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        int cnt = 0;
        for (int k=0;k<n;++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float x2=xyz2[j*3+0];
            float y2=xyz2[j*3+1];
            float z2=xyz2[j*3+2];
            float x1=xyz1[k*3+0];
            float y1=xyz1[k*3+1];
            float z1=xyz1[k*3+2];
    	    float d=max(sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1)),1e-20f);
            if (d<radius) {
                if (cnt==0) { // set ALL indices to k, s.t. if there are less points in ball than nsample, we still have valid (repeating) indices
                    for (int l=0;l<nsample;++l)
                        idx[j*nsample+l] = k;
                }
                idx[j*nsample+cnt] = k;
                cnt+=1;
            }
        }
        pts_cnt[j] = cnt;
    }
}

// input: points (b,n,c), idx (b,m,nsample)
// output: out (b,m,nsample,c)
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
    }
}



// input: grad_out (b,m,nsample,c), idx (b,m,nsample), 
// output: grad_points (b,n,c)
__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m*nsample*batch_index;
    grad_out += m*nsample*c*batch_index;
    grad_points += n*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<m;j+=stride) {
        for (int k=0;k<nsample;++k) {
            int ii = idx[j*nsample+k];
            for (int l=0;l<c;++l) {
                 atomicAdd(&grad_points[ii*c+l], grad_out[j*nsample*c+k*c+l]);
            }
        }
    }
}


// input: points_feat (b,num_point,c), idx (b,num_query,nsample)
// output: out (b,num_query,c), max_idx (b,num_query,c)
__global__ void group_maxpool_gpu(int b, int num_point, int chan, int num_query, int nsample, const float *points, const int *idx, float *out, int *max_idx) {
    int batch_index = blockIdx.x;
    points += num_point*chan*batch_index;
    idx += num_query*nsample*batch_index;
    out += num_query*chan*batch_index;
    max_idx += num_query*chan*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    float temp_feat;
    int one_max_idx;
    float max_feat=-10000.0;


    for (int j=index;j<num_query;j+=stride) {
        for (int l=0;l<chan;++l) {
            max_feat=-10000.0;
            for (int k=0;k<nsample;++k) {
                int ii = idx[j*nsample+k];
                temp_feat = points[ii*chan+l];
                if(temp_feat>max_feat) {max_feat=temp_feat; one_max_idx=ii;}
            }
            out[j*chan+l] = max_feat;
            max_idx[j*chan+l] = one_max_idx;
        } 
    }
}


// input: grad_out (b,num_query,c), max_idx (b,num_query,c),
// output: grad_points (b,num_point,c)
__global__ void group_maxpool_grad_gpu(int b, int num_point, int chan, int num_query, const float *grad_out, const int *max_idx, float *grad_points) {
    int batch_index = blockIdx.x;
    max_idx += num_query*chan*batch_index;
    grad_out += num_query*chan*batch_index;
    grad_points += num_point*chan*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j=index;j<num_query;j+=stride) {
        for (int l=0;l<chan;++l) {
            int ii = max_idx[j*chan+l];
            atomicAdd(&grad_points[ii*chan+l], grad_out[j*chan+l]);
        }
    }
}






// input: k (1), distance matrix dist (b,m,n)
// output: idx (b,m,n), dist_out (b,m,n)
// only the top k results within n are useful
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist+=m*n*batch_index;
    outi+=m*n*batch_index;
    out+=m*n*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // copy from dist to dist_out
    for (int j=index;j<m;j+=stride) {
        for (int s=0;s<n;++s) {
            out[j*n+s] = dist[j*n+s];
            outi[j*n+s] = s;
        }
    }

    float *p_dist;
    for (int j=index;j<m;j+=stride) {
        p_dist = out+j*n;
        // selection sort for the first k elements
        for (int s=0;s<k;++s) {
            int min=s; 
            // find the min
            for (int t=s+1;t<n;++t) {
                if (p_dist[t]<p_dist[min]) {
                    min = t;
                }
            }
            // swap min-th and i-th element
            if (min!=s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j*n+min];
                outi[j*n+min] = outi[j*n+s];
                outi[j*n+s] = tmpi;
            }
        }
    }
}

void queryBallPointLauncher(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    query_ball_point_gpu<<<b,256>>>(b,n,m,radius,nsample,xyz1,xyz2,idx,pts_cnt);
    //cudaDeviceSynchronize();
}
void selectionSortLauncher(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    selection_sort_gpu<<<b,256>>>(b,n,m,k,dist,outi,out); 
    //cudaDeviceSynchronize();
}
void groupPointLauncher(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out){
    group_point_gpu<<<b,256>>>(b,n,c,m,nsample,points,idx,out);
    //cudaDeviceSynchronize();
}
void groupPointGradLauncher(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points){
    group_point_grad_gpu<<<b,256>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //group_point_grad_gpu<<<1,1>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //cudaDeviceSynchronize();
}
void groupMaxpoolLauncher(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out, int *max_idx){
    group_maxpool_gpu<<<b,256>>>(b,n,c,m,nsample,points,idx,out,max_idx);
    //cudaDeviceSynchronize();
}
void groupMaxpoolGradLauncher(int b, int n, int c, int m, const float *grad_out, const int *max_idx, float *grad_points){
    group_maxpool_grad_gpu<<<b,256>>>(b,n,c,m,grad_out,max_idx,grad_points);
    //group_point_grad_gpu<<<1,1>>>(b,n,c,m,nsample,grad_out,idx,grad_points);
    //cudaDeviceSynchronize();
}




